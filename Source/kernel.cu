#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<complex.h>

#include "mandelbrot.h"
#include "bitmap.h"
#include "args.h"

// Used for bitmap to/from complex conversions.
long double step_size;
long double x_min, y_min;

/**
 * @brief Iterates on grid to generate mandelbrot set points
 * 
 * @param grid the grid
 */
// __global__ void _mandelbrot_kernel(unsigned char ** grid, Bitmap *bitmap, int grid_width, int grid_height, int grid_offset_y, int iterations){
__global__ void _mandelbrot_kernel(Rgb ** grid, Bitmap *bitmap, int grid_width, int grid_height, int grid_offset_y, int iterations){

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(; index < grid_width * grid_height; index += stride) {
        int grid_x = index / grid_width;
        int grid_y = grid_offset_y + index % grid_height;

        long double c_real, c_imag;
        _bitmap_to_complex(grid_x, grid_y, &c_real, &c_imag);

        MB_Point point = MB_iterate_mandelbrot(c_real, c_imag, iterations);
        Rgb color = MB_color_of(&point, DIRECT_RGB);

        grid[grid_x][grid_y] = color;
    }
}

/**
 * @brief starts the mandelbrot kernel with @p blocksize threads with each point undergoing @p num_iterations
 * 
 * @param num_iterations number of iterations per point
 * @param block_size number of threads per block
 */
// extern "C" void launch_mandelbrot_kernel(unsigned char ** grid, Bitmap *bitmap, int grid_width, int grid_height, int grid_offset_y, int iterations, int block_size){
extern "C" void launch_mandelbrot_kernel(Rgb ** grid, Bitmap *bitmap, int grid_width, int grid_height, int grid_offset_y, int iterations, int block_size){
    int N = grid_width * grid_height;
    int num_blocks = (N + block_size - 1) / block_size;

    // Launch kernel
    _mandelbrot_kernel<<<num_blocks, block_size>>>(grid, bitmap, grid_width, grid_height, grid_offset_y, args->iterations);
    // Synchronize threads
    hipDeviceSynchronize();
}

extern "C" void cuda_init(int my_rank) {
	int cudaDeviceCount;
	hipError_t cE;
	if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount );
        exit(-1);
    }
    if( (cE = hipSetDevice( my_rank % cudaDeviceCount )) != hipSuccess )
    {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n", my_rank, (my_rank % cudaDeviceCount), cE);
        exit(-1);
    }
}
